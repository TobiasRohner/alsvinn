#include "hip/hip_runtime.h"
/* Copyright (c) 2018 ETH Zurich, Kjetil Olsen Lye
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "alsuq/stats/StructureBasicCUDA.hpp"
#include "alsfvm/volume/volume_foreach.hpp"
#include "alsuq/stats/stats_util.hpp"
namespace alsuq {
namespace stats {


namespace {

//! Computes the structure function for FIXED h
//!
//! The goal is to compute the structure function, then reduce (sum) over space
//! then go on to next h
__global__ void computeStructureBasic(real* output,
    alsfvm::memory::View<const real> input,
    ivec3 directionVector,
    int h,
    int nx, int ny, int nz, int ngx, int ngy, int ngz,
    real p) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index >= nx * ny * nz) {
        return;
    }

    const int x = index % nx;
    const int y = (index / nx) % ny;
    const int z = (index / nx / ny);



    const int xNext = x + h * directionVector.x;
    const int yNext = y + h * directionVector.y;
    const int zNext = z + h * directionVector.z;

    const real u = input.at(x + ngx, y + ngy, z + ngz);
    const real u_h = input.at(xNext % nx + ngx, yNext % ny + ngy, zNext % nz + ngz);

    output[index] = pow(fabs(u - u_h), p);
}
}

StructureBasicCUDA::StructureBasicCUDA(const StatisticsParameters& parameters)
    : StatisticsHelper(parameters),
      direction(parameters.getInteger("direction")),
      p(parameters.getDouble("p")),
      directionVector(make_direction_vector(direction)),
      numberOfH(parameters.getInteger("numberOfH")),
      statisticsName ("structure_basic_" + std::to_string(p))

{

}

std::vector<std::string> StructureBasicCUDA::getStatisticsNames() const {
    return {statisticsName};
}

void StructureBasicCUDA::computeStatistics(const alsfvm::volume::Volume&
    conservedVariables,
    const alsfvm::volume::Volume& extraVariables,
    const alsfvm::grid::Grid& grid,
    const alsfvm::simulator::TimestepInformation& timestepInformation) {
    auto& structure = this->findOrCreateSnapshot(statisticsName,
            timestepInformation,
            conservedVariables, extraVariables,
            numberOfH, 1, 1, "cpu");


    computeStructure(*structure.getVolumes().getConservedVolume(),
        conservedVariables);
    computeStructure(*structure.getVolumes().getExtraVolume(),
        extraVariables);
}

void StructureBasicCUDA::finalizeStatistics() {

}

void StructureBasicCUDA::computeStructure(alsfvm::volume::Volume& output,
    const alsfvm::volume::Volume& input) {

    for (size_t var = 0; var < input.getNumberOfVariables(); ++var) {
        auto inputView = input[var]->getView();
        auto outputView = output[var]->getView();

        const int ngx = input.getNumberOfXGhostCells();
        const int ngy = input.getNumberOfYGhostCells();
        const int ngz = input.getNumberOfZGhostCells();

        const int nx = int(input.getNumberOfXCells()) - 2 * ngx;
        const int ny = int(input.getNumberOfYCells()) - 2 * ngy;
        const int nz = int(input.getNumberOfZCells()) - 2 * ngz;

        const int dimensions = input.getDimensions();

        structureOutput.resize(nx * ny * nz, 0);

        for (int h = 1; h < int(numberOfH); ++h) {
            const int threads = 1024;
            const int size = nx * ny * nz;
            const int blockNumber = (size + threads - 1) / threads;

            computeStructureBasic <<< blockNumber, threads>>>(thrust::raw_pointer_cast(
                    structureOutput.data()), inputView, directionVector,
                h, nx, ny, nz, ngx, ngy, ngz, p);

            real structureResult = thrust::reduce(structureOutput.begin(),
                    structureOutput.end(),
                    0.0, thrust::plus<real>());

            outputView.at(h) += structureResult / (nx * ny * nz);
        }



    }
}
REGISTER_STATISTICS(cuda, structure_basic, StructureBasicCUDA)
}
}
