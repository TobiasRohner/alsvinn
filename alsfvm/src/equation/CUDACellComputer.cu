#include "hip/hip_runtime.h"
#include "alsfvm/equation/CUDACellComputer.hpp"
#include "alsfvm/equation/CPUCellComputer.hpp"
#include "alsfvm/equation/euler/Euler.hpp"
#include "alsfvm/volume/volume_foreach.hpp"
#include "alsfvm/cuda/cuda_utils.hpp"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

namespace alsfvm {
	namespace equation {

		namespace {
			///
			/// Computes the extra variables for each cell.
			///
			template<class Equation>
			__global__ void computeExtraVariablesDevice(typename Equation::ConstViews conservedIn,
				typename Equation::ViewsExtra extra, size_t size) {
				size_t index = blockIdx.x * blockDim.x + threadIdx.x;

				if (index >= size) {
					return;
				}
				auto conservedStruct = Equation::fetchConservedVariables(conservedIn, index);
				auto extraStruct = Equation::computeExtra(conservedStruct);
				Equation::setExtraViewAt(extra, index, extraStruct);
			}


			///
			/// Computes the wave speed for each cell in the given direction.
			/// To get the maximum wavespeed one needs to do a reduction.
			///
			template<class Equation, size_t direction>
			__global__ void computeWaveSpeedDevice(typename Equation::ConstViews conserved, typename Equation::ConstViewsExtra extra,
				size_t size, real* outputPointer) {

				const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
				if (index >= size) {
					return;
				}
				
				outputPointer[index] = Equation::template computeWaveSpeed < direction >(Equation::fetchConservedVariables(conserved, index),
					Equation::fetchExtraVariables(extra, index));
			}

			///
			/// Computes the wave speed for the given direction, then reduces the output (using thrust)
			/// \note We might want to change the thrust implementation to something faster eventually, but we
			/// will do more profiling first.
			///
			template<class Equation, size_t direction>
			real computeWaveSpeedAndReduce(const volume::Volume &conservedVariables,
				const volume::Volume &extraVariables, thrust::device_vector<real>& deviceVector) {
				const size_t size = conservedVariables.getScalarMemoryArea(0)->getSize();
				const size_t blockSize = 1024;

				computeWaveSpeedDevice<Equation, direction> << <(size + blockSize - 1)/blockSize, blockSize>> >(typename Equation::ConstViews(conservedVariables),
					typename Equation::ConstViewsExtra(extraVariables), size, thrust::raw_pointer_cast(&deviceVector[0]));
				CUDA_SAFE_CALL(hipStreamSynchronize(0));
				// For now we simply use thrust to reduce.
				return thrust::reduce(deviceVector.begin(), deviceVector.end(), 0.0, thrust::maximum<real>());
			}


			/// 
			/// Checks for each cell if we obey the constraints given.
			/// One should do a reduction over the results to get if all cells obeys or not.
			///
			template<class Equation>
			__global__ void checkObeysConstraintsDevice(typename Equation::ConstViews conserved, typename Equation::ConstViewsExtra extra,
				size_t size, size_t* outputPointer) {
				size_t index = blockIdx.x * blockDim.x + threadIdx.x;

				if (index >= size) {
					return;
				}
				outputPointer[index] = Equation::obeysConstraints(Equation::fetchConservedVariables(conserved, index),
					Equation::fetchExtraVariables(extra, index));
			}


			///
			/// Checks if each cells obeys the constraints, then reduces the output using thrust.
			///
			template<class Equation>
			bool checkObeysConstraintsAndReduce(const volume::Volume &conservedVariables,
				const volume::Volume &extraVariables, thrust::device_vector<size_t>& deviceVector) {

				const size_t size = conservedVariables.getScalarMemoryArea(0)->getSize();
				const size_t blockSize = 1024;


				checkObeysConstraintsDevice<Equation> << <(size + blockSize - 1) / blockSize, blockSize >> >(typename Equation::ConstViews(conservedVariables),
					typename Equation::ConstViewsExtra(extraVariables),
					size, thrust::raw_pointer_cast(&deviceVector[0]));
				CUDA_SAFE_CALL(hipStreamSynchronize(0));
				// For now we simply use thrust to reduce.
				return thrust::reduce(deviceVector.begin(), deviceVector.end(), 0, thrust::maximum<size_t>());
			}


		}

		template<class Equation>
		void CUDACellComputer<Equation>::computeExtraVariables(const volume::Volume &conservedVariables,
			volume::Volume &extraVariables)
		{
			const size_t size = conservedVariables.getScalarMemoryArea(0)->getSize();
			const size_t blockSize = 1024;


			computeExtraVariablesDevice<Equation> << <(size + blockSize - 1) / blockSize, blockSize >> >(typename Equation::ConstViews(conservedVariables),
				typename Equation::ViewsExtra(extraVariables), size);
		}


		template<class Equation>
		real CUDACellComputer<Equation>::computeMaxWaveSpeed(const volume::Volume& conservedVariables,
			const volume::Volume& extraVariables, size_t direction) {

			// We declare this static to avoid having to reallocate it every time,
			// and to avoid having to expose it in the class interface.
			static thrust::device_vector<real> deviceVector;
			deviceVector.resize(conservedVariables.getScalarMemoryArea(0)->getSize(), 0.0);
			assert(direction < 3);
			if (direction == 0) {
				return computeWaveSpeedAndReduce<Equation, 0>(conservedVariables, extraVariables, deviceVector);
			}
			if (direction == 1) {
				return computeWaveSpeedAndReduce<Equation, 1>(conservedVariables, extraVariables, deviceVector);
			}
			if (direction == 2) {
				return computeWaveSpeedAndReduce<Equation, 2>(conservedVariables, extraVariables, deviceVector);
			}

		}

		/// 
		/// Checks if all the constraints for the equation are met
		///	\param conservedVariables the conserved variables (density, momentum, Energy for Euler)
		/// \param extraVariables the extra variables (pressure and velocity for Euler)
		/// \return true if it obeys the constraints, false otherwise
		/// \todo Tidy up the way we check for nan and inf
		///
		template<class Equation>
		bool CUDACellComputer<Equation>::obeysConstraints(const volume::Volume& conservedVariables,
			const volume::Volume& extraVariables) {
			static thrust::device_vector<size_t> deviceVector;
			deviceVector.resize(conservedVariables.getScalarMemoryArea(0)->getSize(), 0);

			return checkObeysConstraintsAndReduce<Equation>(conservedVariables, extraVariables, deviceVector);
		}

		template<class Equation>
		void CUDACellComputer<Equation>::computeFromPrimitive(const volume::Volume &primitiveVariables,
			volume::Volume &conservedVariables,
			volume::Volume &extraVariables)
		{
			THROW("Unsupported operation. We do not support calculating primitive variables on the GPU, this should be done on the CPU for now.")
		}

		template class CUDACellComputer < euler::Euler > ;
	}
}


