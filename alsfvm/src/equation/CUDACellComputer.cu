#include "hip/hip_runtime.h"
#include "alsfvm/equation/CUDACellComputer.hpp"
#include "alsfvm/equation/CPUCellComputer.hpp"
#include "alsfvm/equation/euler/Euler.hpp"

#include "alsfvm/cuda/cuda_utils.hpp"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

namespace alsfvm {
	namespace equation {

		namespace {
			///
			/// Computes the extra variables for each cell.
			///
			template<class Equation>
			__global__ void computeExtraVariablesDevice(Equation eq, typename Equation::ConstViews conservedIn,
				typename Equation::ViewsExtra extra, size_t size) {
				size_t index = blockIdx.x * blockDim.x + threadIdx.x;

				if (index >= size) {
					return;
				}
				typename Equation::ConservedVariables conservedStruct = eq.fetchConservedVariables(conservedIn, index);
				typename Equation::ExtraVariables extraStruct = eq.computeExtra(conservedStruct);
				eq.setExtraViewAt(extra, index, extraStruct);
			}


			///
			/// Computes the wave speed for each cell in the given direction.
			/// To get the maximum wavespeed one needs to do a reduction.
			///
			template<class Equation, size_t direction>
			__global__ void computeWaveSpeedDevice(Equation eq, typename Equation::ConstViews conserved, typename Equation::ConstViewsExtra extra,
				size_t size, real* outputPointer) {

				const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
				if (index >= size) {
					return;
				}
				
				outputPointer[index] = eq.template computeWaveSpeed < direction >(eq.fetchConservedVariables(conserved, index),
					eq.fetchExtraVariables(extra, index));
			}

			///
			/// Computes the wave speed for the given direction, then reduces the output (using thrust)
			/// \note We might want to change the thrust implementation to something faster eventually, but we
			/// will do more profiling first.
			///
			template<class Equation, size_t direction>
			real computeWaveSpeedAndReduce(const Equation& eq, const volume::Volume &conservedVariables,
				const volume::Volume &extraVariables, thrust::device_vector<real>& deviceVector) {
				const size_t size = conservedVariables.getScalarMemoryArea(0)->getSize();
				const size_t blockSize = 1024;

				computeWaveSpeedDevice<Equation, direction> << <(size + blockSize - 1)/blockSize, blockSize>> >(eq, typename Equation::ConstViews(conservedVariables),
					typename Equation::ConstViewsExtra(extraVariables), size, thrust::raw_pointer_cast(&deviceVector[0]));
				CUDA_SAFE_CALL(hipStreamSynchronize(0));
				// For now we simply use thrust to reduce.
				return thrust::reduce(deviceVector.begin(), deviceVector.end(), 0.0, thrust::maximum<real>());
			}


			/// 
			/// Checks for each cell if we obey the constraints given.
			/// One should do a reduction over the results to get if all cells obeys or not.
			///
			template<class Equation>
			__global__ void checkObeysConstraintsDevice(Equation eq, typename Equation::ConstViews conserved, typename Equation::ConstViewsExtra extra,
				size_t size, size_t* outputPointer) {
				size_t index = blockIdx.x * blockDim.x + threadIdx.x;

				if (index >= size) {
					return;
				}
				outputPointer[index] = eq.obeysConstraints(eq.fetchConservedVariables(conserved, index),
					eq.fetchExtraVariables(extra, index));
			}


			///
			/// Checks if each cells obeys the constraints, then reduces the output using thrust.
			///
			template<class Equation>
			bool checkObeysConstraintsAndReduce(const Equation& equation, const volume::Volume &conservedVariables,
				const volume::Volume &extraVariables, thrust::device_vector<size_t>& deviceVector) {

				const size_t size = conservedVariables.getScalarMemoryArea(0)->getSize();
				const size_t blockSize = 1024;


				checkObeysConstraintsDevice<Equation> << <(size + blockSize - 1) / blockSize, blockSize >> >(equation, typename Equation::ConstViews(conservedVariables),
					typename Equation::ConstViewsExtra(extraVariables),
					size, thrust::raw_pointer_cast(&deviceVector[0]));
				CUDA_SAFE_CALL(hipStreamSynchronize(0));
				// For now we simply use thrust to reduce.
				return bool(thrust::reduce(deviceVector.begin(), deviceVector.end(), 0, thrust::maximum<size_t>()));
			}


		}

		template<class Equation>
		void CUDACellComputer<Equation>::computeExtraVariables(const volume::Volume &conservedVariables,
			volume::Volume &extraVariables)
		{
			const size_t size = conservedVariables.getScalarMemoryArea(0)->getSize();
			const size_t blockSize = 1024;


			computeExtraVariablesDevice<Equation> << <(size + blockSize - 1) / blockSize, blockSize >> >(equation, typename Equation::ConstViews(conservedVariables),
				typename Equation::ViewsExtra(extraVariables), size);
		}


		template<class Equation>
		real CUDACellComputer<Equation>::computeMaxWaveSpeed(const volume::Volume& conservedVariables,
			const volume::Volume& extraVariables, size_t direction) {

			// We declare this static to avoid having to reallocate it every time,
			// and to avoid having to expose it in the class interface.
			static thrust::device_vector<real> deviceVector;
			deviceVector.resize(conservedVariables.getScalarMemoryArea(0)->getSize(), 0.0);
			assert(direction < 3);
			if (direction == 0) {
				return computeWaveSpeedAndReduce<Equation, 0>(equation, conservedVariables, extraVariables, deviceVector);
			}
			if (direction == 1) {
				return computeWaveSpeedAndReduce<Equation, 1>(equation, conservedVariables, extraVariables, deviceVector);
			}
			if (direction == 2) {
                return computeWaveSpeedAndReduce<Equation, 2>(equation, conservedVariables, extraVariables, deviceVector);
			}
            THROW("Unknown direction: " << direction);

		}

		/// 
		/// Checks if all the constraints for the equation are met
		///	\param conservedVariables the conserved variables (density, momentum, Energy for Euler)
		/// \param extraVariables the extra variables (pressure and velocity for Euler)
		/// \return true if it obeys the constraints, false otherwise
		/// \todo Tidy up the way we check for nan and inf
		///
		template<class Equation>
		bool CUDACellComputer<Equation>::obeysConstraints(const volume::Volume& conservedVariables,
			const volume::Volume& extraVariables) {
			static thrust::device_vector<size_t> deviceVector;
			deviceVector.resize(conservedVariables.getScalarMemoryArea(0)->getSize(), 0);

			return checkObeysConstraintsAndReduce<Equation>(equation, conservedVariables, extraVariables, deviceVector);
		}

		template<class Equation>
		void CUDACellComputer<Equation>::computeFromPrimitive(const volume::Volume &primitiveVariables,
			volume::Volume &conservedVariables,
			volume::Volume &extraVariables)
		{
			THROW("Unsupported operation. We do not support calculating primitive variables on the GPU, this should be done on the CPU for now.")
		}

        template<class Equation>
        CUDACellComputer<Equation>::CUDACellComputer(simulator::SimulatorParameters& simulatorParameters) 
            : equation(static_cast<typename Equation::Parameters&>(simulatorParameters.getEquationParameters()))
        {

        }
		template class CUDACellComputer < euler::Euler > ;
	}
}


