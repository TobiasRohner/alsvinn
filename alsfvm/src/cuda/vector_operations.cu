#include "hip/hip_runtime.h"
/* Copyright (c) 2018 ETH Zurich, Kjetil Olsen Lye
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "alsfvm/cuda/vector_operations.hpp"
#include "alsfvm/types.hpp"
#include "alsfvm/cuda/cuda_utils.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

namespace {
template<class T>
__global__ void addKernel(T* result, const T* a, const T* b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

template<class T>
__global__ void multiplyKernel(T* result, const T* a, const T* b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] * b[index];
    }
}

template<class T>
__global__ void divideKernel(T* result, const T* a, const T* b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] / b[index];
    }
}

template<class T>
__global__ void subtractKernel(T* result, const T* a, const T* b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] - b[index];
    }
}

template<class T>
__global__ void addKernel(T* result, const T* a, T b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] + b;
    }
}

template<class T>
__global__ void multiplyKernel(T* result, const T* a, T b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] * b;
    }
}

template<class T>
__global__ void divideKernel(T* result, const T* a, T b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] / b;
    }
}

template<class T>
__global__ void subtractKernel(T* result, const T* a, T b, size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        result[index] = a[index] - b;
    }
}

template<class T>
__global__ void linear_combination_device(T a1, T* v1,
    T a2, const T* v2,
    T a3, const T* v3,
    T a4, const T* v4,
    T a5, const T* v5,
    size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        v1[index] = a1 * v1[index] + a2 * v2[index] + a3 * v3[index] + a4 * v4[index] +
            a5 * v5[index];
    }
}


template<class T>
__global__ void add_power_device(T* out, const T* a, double power,
    size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        out[index] += pow(a[index], power);
    }
}


template<class T>
__global__ void add_power_device(T* out, const T* a, double power, double factor,
    size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        out[index] += factor * pow(a[index], power);
    }
}


template<class T>
__global__ void subtract_power_device(T* out, const T* a, double power,
    size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        out[index] -= pow(a[index], power);
    }
}

template<class T>
__global__ void compute_total_variation_device(T* out, const T* data,
    alsfvm::ivec3 start, alsfvm::ivec3 end, int nx, int ny, int nz, int p) {

    const auto coordinates = alsfvm::cuda::getCoordinates(threadIdx, blockIdx,
            blockDim,
            (end - start).x,
            (end - start).y,
            (end - start).z,
            start);


    if (coordinates.x < 0) {
        return;
    }

    const int x = coordinates.x;
    const int y = coordinates.y;
    const int z = coordinates.z;

    const size_t index = z * nx * ny + y * nx + x;
    const size_t indexXLeft = z * nx * ny + y * nx + (x - 1);

    const size_t yBottom = ny > 0 ? y - 1 : 0;

    const size_t indexYLeft = z * nx * ny + yBottom * nx + x;


    out[index] = pow(sqrt(pow(data[index]
                    - data[indexXLeft], 2) + pow( data[index]
                    - data[indexYLeft], 2)), p);

}


template<class T>
__global__ void compute_total_variation_device(T* out, const T* data,
    alsfvm::ivec3 start, alsfvm::ivec3 end, int nx, int ny, int nz,
    size_t direction, int p) {

    const auto directionVector = alsfvm::make_direction_vector(direction);
    const auto coordinates = alsfvm::cuda::getCoordinates(threadIdx, blockIdx,
            blockDim,
            (end - start - directionVector).x,
            (end - start - directionVector).y,
            (end - start - directionVector).z,
            start + directionVector);


    if (coordinates.x < 0) {
        return;
    }

    const int x = coordinates.x;
    const int y = coordinates.y;
    const int z = coordinates.z;

    const size_t index = z * nx * ny + y * nx + x;


    const auto coordinatesLeft = coordinates - directionVector;


    const size_t indexLeft = coordinatesLeft.z * nx * ny + coordinatesLeft.y * nx +
        coordinatesLeft.x;

    out[index] = pow(fabs(data[index] - data[indexLeft]), p);
}
}

// Since we use templates, we must instatiate.
#define INSTANTIATE_VECTOR_OPERATION(x)\
    template void x<real>(real*, const real*, const real*, size_t);

#define INSTANTIATE_VECTOR_SCALAR_OPERATION(x)\
    template void x<real>(real*, const real*, const real, size_t);



namespace alsfvm {
namespace cuda {

///
/// Adds a and b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a or b). Must have length size (in T)
/// \param a must have length size (in T)
/// \param b must have length size (in T)
///
template<class T>
void add(T* result, const T* a, const T* b, size_t size) {
    const size_t threadCount = 1024;
    addKernel << < (size + threadCount - 1) / threadCount, threadCount >> > (result,
            a, b, size);
}

///
/// Multiplies a and b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a or b). Must have length size (in T)
/// \param a must have length size (in T)
/// \param b must have length size (in T)
///
template<class T>
void multiply(T* result, const T* a, const T* b, size_t size) {
    const size_t threadCount = 1024;
    multiplyKernel << < (size + threadCount - 1) / threadCount,
                   threadCount >> > (result, a, b, size);
}

///
/// Subtracts a and b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a or b). Must have length size (in T)
/// \param a must have length size (in T)
/// \param b must have length size (in T)
///
template<class T>
void subtract(T* result, const T* a, const T* b, size_t size) {
    const size_t threadCount = 1024;
    subtractKernel << < (size + threadCount - 1) / threadCount,
                   threadCount >> > (result, a, b, size);
}
///
/// Divides a and b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a or b). Must have length size (in T)
/// \param a must have length size (in T)
/// \param b must have length size (in T)
///
template<class T>
void divide(T* result, const T* a, const T* b, size_t size) {
    const size_t threadCount = 1024;
    divideKernel << < (size + threadCount - 1) / threadCount,
                 threadCount >> > (result, a, b, size);
}

///
/// Adds scalar to each component of b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a ). Must have length size (in T)
/// \param a must have length size (in T)
/// \param scalar the scalar
///
template<class T>
void add(T* result, const T* a, T scalar, size_t size) {
    const size_t threadCount = 1024;
    addKernel << < (size + threadCount - 1) / threadCount, threadCount >> > (result,
            a, scalar, size);
}

///
/// Multiplies scalar to each component of b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a ). Must have length size (in T)
/// \param a must have length size (in T)
/// \param scalar the scalar
///
template<class T>
void multiply(T* result, const T* a, T scalar, size_t size) {
    const size_t threadCount = 1024;
    multiplyKernel << < (size + threadCount - 1) / threadCount,
                   threadCount >> > (result, a, scalar, size);
}

///
/// Subtracts scalar from each component of b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a ). Must have length size (in T)
/// \param a must have length size (in T)
/// \param scalar the scalar
///
template<class T>
void subtract(T* result, const T* a, T scalar, size_t size) {
    const size_t threadCount = 1024;
    subtractKernel << < (size + threadCount - 1) / threadCount,
                   threadCount >> > (result, a, scalar, size);
}

///
/// Divides scalar from each component of b and stores the result to result
/// \param result the device memory to write to (can be the same
///               as a ). Must have length size (in T)
/// \param a must have length size (in T)
/// \param scalar the scalar
///
template<class T>
void divide(T* result, const T* a, T scalar, size_t size) {
    const size_t threadCount = 1024;
    divideKernel <<< (size + threadCount - 1) / threadCount,
                 threadCount >>> (result, a, scalar, size);
}

template<class T>
void add_linear_combination(T a1, T* v1,
    T a2, const T* v2,
    T a3, const T* v3,
    T a4, const T* v4,
    T a5, const T* v5,
    size_t size) {
    const size_t threadCount = 1024;
    linear_combination_device << < (size + threadCount - 1) / threadCount,
                              threadCount >> > (a1, v1, a2, v2, a3, v3, a4, v4, a5, v5, size);
}

template<class T>
void add_power(T* a, const T* b, double power, size_t size) {
    const size_t threadCount = 1024;
    add_power_device << < (size + threadCount - 1) / threadCount,
                     threadCount >> > (a, b, power, size);
}


template<class T>
void add_power(T* a, const T* b, double power, double factor, size_t size) {
    const size_t threadCount = 1024;
    add_power_device << < (size + threadCount - 1) / threadCount,
                     threadCount >> > (a, b, power, factor, size);
}



template<class T>
void subtract_power(T* a, const T* b, double power, size_t size) {
    const size_t threadCount = 1024;
    subtract_power_device << < (size + threadCount - 1) / threadCount,
                          threadCount >> > (a, b, power, size);
}

template<class T>
T compute_total_variation(const T* a, size_t nx, size_t ny, size_t nz, int p,
    const ivec3& start, const ivec3& end) {
    thrust::device_vector<T> buffer(nx * ny * nz, 0);

    if (nz > 1) {
        THROW("Only supported for 2d and 1d");
    }

    auto launchParameters = cuda::makeKernelLaunchParameters(start + ivec3{1, ny > 1, nz > 1},
            end,
            1024);


    compute_total_variation_device <<< std::get<0>(launchParameters), 1024>>>
    (thrust::raw_pointer_cast(buffer.data()),
        a, start + ivec3{1, ny > 1, nz > 1}, end, nx, ny, nz, p);

    return thrust::reduce(buffer.begin(), buffer.end());




}


template<class T>
T compute_total_variation(const T* a, size_t nx, size_t ny, size_t nz,
    size_t direction, int p,
    const ivec3& start, const ivec3& end) {
    thrust::device_vector<T> buffer(nx * ny * nz, 0);

    if (nz > 1) {
        THROW("Only supported for 2d and 1d");
    }

    auto directionVector = make_direction_vector(direction);
    auto launchParameters = cuda::makeKernelLaunchParameters(
            start + directionVector,
            end,
            1024);


    compute_total_variation_device <<< std::get<0>(launchParameters), 1024>>>
    (thrust::raw_pointer_cast(buffer.data()),
        a, start, end, nx, ny, nz, direction, p);

    return thrust::reduce(buffer.begin(), buffer.end());



}

INSTANTIATE_VECTOR_OPERATION(add)
INSTANTIATE_VECTOR_OPERATION(subtract)
INSTANTIATE_VECTOR_OPERATION(multiply)
INSTANTIATE_VECTOR_OPERATION(divide)

INSTANTIATE_VECTOR_SCALAR_OPERATION(add)
INSTANTIATE_VECTOR_SCALAR_OPERATION(subtract)
INSTANTIATE_VECTOR_SCALAR_OPERATION(multiply)
INSTANTIATE_VECTOR_SCALAR_OPERATION(divide)

template void add_linear_combination<real>(real a1, real* v1, real a2,
    const real* v2,
    real a3, const real* v3,
    real a4, const real* v4,
    real a5, const real* v5,
    size_t size);

template void add_power<real>(real* a, const real* b, double power,
    size_t size);


template void add_power<real>(real* a, const real* b, double power, real factor,
    size_t size);

template void subtract_power<real>(real* a, const real* b, double power,
    size_t size);

template real compute_total_variation<real>(const real* a, size_t nx, size_t ny,
    size_t nz, int p, const ivec3& start, const ivec3& end);
template real compute_total_variation<real>(const real* a, size_t nx, size_t ny,
    size_t nz, size_t direction, int p, const ivec3& start, const ivec3& end);


}
}
