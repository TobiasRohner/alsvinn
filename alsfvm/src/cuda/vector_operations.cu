#include "hip/hip_runtime.h"
#include "alsfvm/cuda/vector_operations.hpp"
#include "alsfvm/types.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace {
	template<class T>
	__global__ void addKernel(T* result, const T* a, const T* b, size_t size) {
		size_t index = blockIdx.x*blockDim.x + threadIdx.x;

		if (index < size) {
			result[index] = a[index] + b[index];
		}
	}

	template<class T>
	__global__ void multiplyKernel(T* result, const T* a, const T* b, size_t size) {
		size_t index = blockIdx.x*blockDim.x + threadIdx.x;

		if (index < size) {
			result[index] = a[index] * b[index];
		}
	}

	template<class T>
	__global__ void divideKernel(T* result, const T* a, const T* b, size_t size) {
		size_t index = blockIdx.x*blockDim.x + threadIdx.x;

		if (index < size) {
			result[index] = a[index]/b[index];
		}
	}

	template<class T>
	__global__ void subtractKernel(T* result, const T* a, const T* b, size_t size) {
		size_t index = blockIdx.x*blockDim.x + threadIdx.x;

		if (index < size) {
			result[index] = a[index] - b[index];
		}
	}
}

// Since we use templates, we must instatiate.
#define INSTANTIATE_VECTOR_OPERATION(x)\
	template void x<real>(real*, const real*, const real*, size_t);


namespace alsfvm {
	namespace cuda {

		///
		/// Adds a and b and stores the result to result
		/// \param result the device memory to write to (can be the same
		///               as a or b). Must have length size (in T)
		/// \param a must have length size (in T)
		/// \param b must have length size (in T)
		///
		template<class T>
		void add(T* result, const T* a, const T* b, size_t size) {
			const size_t threadCount = 1024;
			addKernel <<<(size + threadCount - 1), threadCount >>>(result, a, b, size);
		}

		///
		/// Multiplies a and b and stores the result to result
		/// \param result the device memory to write to (can be the same
		///               as a or b). Must have length size (in T)
		/// \param a must have length size (in T)
		/// \param b must have length size (in T)
		///
		template<class T>
		void multiply(T* result, const T* a, const T* b, size_t size) {
			const size_t threadCount = 1024;
			multiplyKernel <<<(size + threadCount - 1), threadCount >>>(result, a, b, size);
		}

		///
		/// Subtracts a and b and stores the result to result
		/// \param result the device memory to write to (can be the same
		///               as a or b). Must have length size (in T)
		/// \param a must have length size (in T)
		/// \param b must have length size (in T)
		///
		template<class T>
		void subtract(T* result, const T* a, const T* b, size_t size) {
			const size_t threadCount = 1024;
			subtractKernel <<<(size + threadCount - 1), threadCount >>>(result, a, b, size);
		}
		///
		/// Divides a and b and stores the result to result
		/// \param result the device memory to write to (can be the same
		///               as a or b). Must have length size (in T)
		/// \param a must have length size (in T)
		/// \param b must have length size (in T)
		///
		template<class T>
		void divide(T* result, const T* a, const T* b, size_t size) {
			const size_t threadCount = 1024;
			divideKernel <<<(size + threadCount - 1), threadCount >>>(result, a, b, size);
		}

		INSTANTIATE_VECTOR_OPERATION(add)
		INSTANTIATE_VECTOR_OPERATION(subtract)
		INSTANTIATE_VECTOR_OPERATION(multiply)
		INSTANTIATE_VECTOR_OPERATION(divide)
	}
}
