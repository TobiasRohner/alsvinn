#include "hip/hip_runtime.h"
#include "alsfvm/reconstruction/ENOCUDA.hpp"
#include "alsutils/error/Exception.hpp"
#include <cassert>
#include <cmath>
#include "alsfvm/reconstruction/ENOCoefficients.hpp"
#include <iostream>

#include <fstream>
#include "alsfvm/equation/equation_list.hpp"
#include "alsfvm/cuda/cuda_utils.hpp"

namespace alsfvm { namespace reconstruction {



template<class Equation, int order>
__global__ void performEnoReconstructionKernel(typename Equation::ConstViews input,
                                               typename Equation::Views left,
                                               typename Equation::Views right,
                                               const gpu_array<real*, order - 1> dividedDifferencesPointers,
                                               const gpu_array<gpu_array<real, order>, order + 1> coefficients,
                                               int numberOfXCells,
                                               int numberOfYCells,
                                               int numberOfZCells,
                                               ivec3 directionVector
                                               ) {


    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    const size_t xInternalFormat = index % numberOfXCells;
    const size_t yInternalFormat = (index / numberOfXCells) % numberOfYCells;
    const size_t zInternalFormat = (index) / (numberOfXCells * numberOfYCells);

    if (xInternalFormat >= numberOfXCells || yInternalFormat >= numberOfYCells
            || zInternalFormat >= numberOfZCells) {
        return;
    }

    const int x = xInternalFormat + (order - 1) * directionVector[0];
    const int y = yInternalFormat + (order - 1) * directionVector[1];
    const int z = zInternalFormat + (order - 1) * directionVector[2];


    const size_t indexRight = input.index(x, y, z);


    const size_t indexLeft = input.index( (x - directionVector.x),
                                          (y - directionVector.y),
                                          (z - directionVector.z));

    // First we determine the shift
    // We do this by looping through the levels of the divided
    // differences, and each time we go left, we increment the shift.
    int shift = 0;
    for (int level = 0; level < order - 1; level++) {
        real dividedDifferenceRight = dividedDifferencesPointers[level][indexRight];
        real dividedDifferenceLeft = dividedDifferencesPointers[level][indexLeft];

        if (fabs(dividedDifferenceLeft) < fabs(dividedDifferenceRight)) {
            // Now we choose the left stencil
            shift++;
        }
    }

    // Now we have the stencil enabled. We need to find the correct
    // coefficients.

    auto coefficientsRight = coefficients[shift + 1];
    auto coefficientsLeft  = coefficients[shift];
    typename Equation::ConservedVariables leftValue;
    typename Equation::ConservedVariables rightValue;
    for (int j = 0; j < order; j++) {

        const size_t index = input.index((x - (shift - j)*directionVector.x),
                                         (y - (shift - j)*directionVector.y),
                                         (z - (shift - j)*directionVector.z));


        const auto value = Equation::fetchConservedVariables(input, index);
        leftValue = leftValue + coefficientsLeft[j] * value;
        rightValue = rightValue + coefficientsRight[j] * value;

    }

    Equation::setViewAt(left, indexRight, leftValue);
    Equation::setViewAt(right, indexRight, rightValue);
}

template<class Equation, int order>
ENOCUDA<Equation, order>::ENOCUDA(alsfvm::shared_ptr<memory::MemoryFactory> &memoryFactory,
                      size_t nx, size_t ny, size_t nz)
{
	size_t ghostX = getNumberOfGhostCells();
	size_t ghostY = ny > 1 ? getNumberOfGhostCells() : 0;
	size_t ghostZ = nz > 1 ? getNumberOfGhostCells() : 0;
    for(size_t i = 0; i < dividedDifferences.size(); i++) {
        dividedDifferences[i] = memoryFactory->createScalarMemory(nx + 2*ghostX, ny + 2*ghostY, nz + 2*ghostZ);
        dividedDifferences[i]->makeZero();
    }
}

template<class Equation, int order>
void ENOCUDA<Equation, order>::performReconstruction(const volume::Volume &inputVariables,
	size_t direction,
	size_t indicatorVariable,
	volume::Volume &leftOut,
	volume::Volume &rightOut)
{
	// We often do compute order-1.
	static_assert(order > 0, "Can not do ENO reconstruction of order 0.");

	if (direction > 2) {
		THROW("Direction can only be 0, 1 or 2, was given: " << direction);
	}
	const ivec3 directionVector(direction == 0, direction == 1, direction == 2);

	// make divided differences

	computeDividedDifferences(*inputVariables.getScalarMemoryArea(indicatorVariable),
		directionVector,
		1,
		*dividedDifferences[0]);

	for (size_t i = 1; i < order - 1; i++) {
		computeDividedDifferences(*dividedDifferences[i - 1],
			directionVector, i + 1, *dividedDifferences[i]);
	}

	// done computing divided differences

	// Now we go on to do the actual reconstruction, choosing the stencil for
	// each point.
	const size_t nx = inputVariables.getTotalNumberOfXCells();
	const size_t ny = inputVariables.getTotalNumberOfYCells();
	const size_t nz = inputVariables.getTotalNumberOfZCells();

	// Sanity check, we need at least ONE point in the interior.
	assert(int(nx) > 2 * directionVector.x * order);
	assert((directionVector.y == 0u) || (int(ny) > 2 * directionVector.y * order));
	assert((directionVector.z == 0u) || (int(nz) > 2 * directionVector.z * order));



    gpu_array<real*, order - 1> dividedDifferencesPointers;

	for (size_t i = 0; i < order - 1; i++) {
		dividedDifferencesPointers[i] = dividedDifferences[i]->getPointer();
	}

    gpu_array<gpu_array<real, order>, order + 1> coefficients;
    for (size_t i = 0; i < order + 1; ++i) {
        for (size_t j = 0; j < order; ++j) {
            coefficients[i][j] = ENOCoeffiecients<order>::coefficients[i][j];
        }
    }

    const ivec3 start = (order - 1) * directionVector;
    const ivec3 end = ivec3(nx, ny, nz) - (order-1) * directionVector;



    const int blockSize = 512;

    const ivec3 numberOfCellsPerDimension = end - start;

    const size_t totalNumberOfCells = size_t(numberOfCellsPerDimension.x) *
            size_t(numberOfCellsPerDimension.y) *
            size_t(numberOfCellsPerDimension.z);

    const int gridSize = (totalNumberOfCells + blockSize - 1 )/ blockSize;

    typename Equation::Views viewLeft(leftOut);
    typename Equation::Views viewRight(rightOut);
    typename Equation::ConstViews viewInput(inputVariables);
#ifndef NDEBUG
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipPeekAtLastError());
#endif
    performEnoReconstructionKernel<Equation, order><<<gridSize, blockSize>>>(viewInput,
                                                            viewLeft,
                                                            viewRight,
                                                            dividedDifferencesPointers,
                                                            coefficients,
                                                            numberOfCellsPerDimension.x,
                                                            numberOfCellsPerDimension.y,
                                                            numberOfCellsPerDimension.z,
                                                            directionVector);
#ifndef NDEBUG
    CUDA_SAFE_CALL(hipPeekAtLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());
#endif
}


template<class Equation, int order>
size_t ENOCUDA<Equation, order>::getNumberOfGhostCells()
{
    return order;
}

__global__ void computeDividedDifferencesKernel(real* output,
                                                const real* input,
                                                size_t numberOfXCells, // total number of
                                                size_t numberOfYCells, // cells minus ghost cells
                                                size_t numberOfZCells, //
                                                size_t nx, // The total number of cells
                                                size_t ny, // in each
                                                size_t nz, // direction
                                                ivec3 direction,
                                                size_t level
                                                ) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    const size_t xInternalFormat = index % numberOfXCells;
    const size_t yInternalFormat = (index / numberOfXCells) % numberOfYCells;
    const size_t zInternalFormat = (index) / (numberOfXCells * numberOfYCells);

    if (xInternalFormat >= numberOfXCells || yInternalFormat >= numberOfYCells || zInternalFormat >= numberOfZCells) {
        return;
    }
    const int x = xInternalFormat + (level) * direction[0];
    const int y = yInternalFormat + (level) * direction[1];
    const int z = zInternalFormat + (level) * direction[2];


    const int indexRight = z*nx*ny + y * nx + x;
    const int indexLeft = (z - direction.z) * nx * ny
            + (y - direction.y) * nx
            + (x - direction.x);

    output[indexLeft] = input[indexRight] - input[indexLeft];

}

template<class Equation, int order>
void ENOCUDA<Equation, order>::computeDividedDifferences(const memory::Memory<real>& input,
                                              const ivec3& direction,
                                              size_t level,
                                              memory::Memory<real>& output)
{


    const int nx = input.getSizeX();
    const int ny = input.getSizeY();
    const int nz = input.getSizeZ();

    // Sanity check, we need at least ONE point in the interior.
    assert(nx > 2*direction.x * level);
    assert(ny > 2*direction.y * level);
    assert(nz > 2*direction.z * level);

    const ivec3 start = int(level) * direction;
    const ivec3 end = ivec3(nx, ny, nz) - int(level) * direction;

    const real* pointerIn = input.getPointer();

    real* pointerOut = output.getPointer();


    const size_t blockSize = 1024;

    const ivec3 numberOfCellsPerDimension = end - start;

    const size_t totalNumberOfCells = size_t(numberOfCellsPerDimension.x) *
            size_t(numberOfCellsPerDimension.y) *
            size_t(numberOfCellsPerDimension.z);

    const size_t gridSize = (totalNumberOfCells + blockSize -1 )/ blockSize;

    computeDividedDifferencesKernel<<<gridSize, blockSize>>>(pointerOut, pointerIn,
                                                             numberOfCellsPerDimension.x,
                                                             numberOfCellsPerDimension.y,
                                                             numberOfCellsPerDimension.z,
                                                             nx, ny, nz,
                                                             direction,
                                                             level);
}

template class ENOCUDA<alsfvm::equation::euler::Euler<1>, 2>;
template class ENOCUDA<alsfvm::equation::euler::Euler<1>, 3>;
template class ENOCUDA<alsfvm::equation::euler::Euler<1>, 4>;

template class ENOCUDA<alsfvm::equation::euler::Euler<2>, 2>;
template class ENOCUDA<alsfvm::equation::euler::Euler<2>, 3>;
template class ENOCUDA<alsfvm::equation::euler::Euler<2>, 4>;

template class ENOCUDA<alsfvm::equation::euler::Euler<3>, 2>;
template class ENOCUDA<alsfvm::equation::euler::Euler<3>, 3>;
template class ENOCUDA<alsfvm::equation::euler::Euler<3>, 4>;

template class ENOCUDA<alsfvm::equation::burgers::Burgers, 2>;
template class ENOCUDA<alsfvm::equation::burgers::Burgers, 3>;
template class ENOCUDA<alsfvm::equation::burgers::Burgers, 4>;

}
}
