#include "hip/hip_runtime.h"
#include "alsfvm/diffusion/TecnoDiffusionCUDA.hpp"
#include "alsfvm/equation/burgers/Burgers.hpp"
#include "alsfvm/diffusion/RoeMatrix.hpp"
#include <iostream>

namespace alsfvm {
    namespace diffusion {
        namespace {

            template<class Equation>
            __global__ void computeEntropyVariables(Equation equation, typename Equation::Views output, typename Equation::ConstViews input) {
                const size_t index = threadIdx.x + blockIdx.x * blockDim.x;

                const size_t nx = input.get(0).getNumberOfXCells();
                const size_t ny = input.get(0).getNumberOfYCells();
                const size_t nz = input.get(0).getNumberOfZCells();

                
                
                if (index >= nx*ny*nz) {
                    return;
                }

                auto in = equation.fetchConservedVariables(input, index);


                auto entropyVariables = equation.computeEntropyVariablesMultipliedByEigenVectorMatrix(in);

                equation.setViewAt(output, index, entropyVariables);
            }

            template<class Equation, class DiffusionMatrix>
            __global__ void multiplyDiffusionMatrix(Equation equation, typename Equation::Views output, 
                typename Equation::ConstViews leftView,
                typename Equation::ConstViews rightView,
                typename Equation::ConstViews conservedView,
                int numberOfXCells,
                int numberOfYCells,
                int numberOfZCells,
                ivec3 directionVector
                ) {


                const int index = threadIdx.x + blockIdx.x * blockDim.x;

                const size_t xInternalFormat = index % numberOfXCells;
                const size_t yInternalFormat = (index / numberOfXCells) % numberOfYCells;
                const size_t zInternalFormat = (index) / (numberOfXCells * numberOfYCells);

                if (xInternalFormat >= numberOfXCells || yInternalFormat >= numberOfYCells
                    || zInternalFormat >= numberOfZCells) {
                    return;
                }

                const int x = xInternalFormat + directionVector[0];
                const int y = yInternalFormat + directionVector[1];
                const int z = zInternalFormat + directionVector[2];

                const size_t rightIndex = output.index(x + directionVector[0], 
                    y + directionVector[1], 
                    z + directionVector[2]);

                const size_t leftIndex = output.index(x - directionVector[0],
                    y - directionVector[1],
                    z - directionVector[2]);

                auto diffusion = [&](size_t left, size_t right) {
                    auto leftValues = Equation::fetchConservedVariables(rightView, left);
                    auto rightValues = Equation::fetchConservedVariables(leftView, right);

                    auto conservedValues = Equation::fetchConservedVariables(conservedView, right);

                    DiffusionMatrix matrix(equation, conservedValues);

                    return 0.5*(equation.computeEigenVectorMatrix(conservedValues) * (matrix * (leftValues - rightValues)));
                };


                Equation::addToViewAt(output, index, diffusion(index, rightIndex) - diffusion(leftIndex, index));

            }

        }


        template<class Equation, class DiffusionMatrix>
        TecnoDiffusionCUDA<Equation, DiffusionMatrix>::TecnoDiffusionCUDA(volume::VolumeFactory& volumeFactory,
            alsfvm::shared_ptr<reconstruction::Reconstruction> reconstruction,
            const simulator::SimulatorParameters& simulatorParameters)

            :
            volumeFactory(volumeFactory),
            reconstruction(reconstruction),
            equation(static_cast<const typename Equation::Parameters&>(simulatorParameters.getEquationParameters()))
        {
            // empty
        }

        template<class Equation, class DiffusionMatrix>
        void TecnoDiffusionCUDA<Equation, DiffusionMatrix>::applyDiffusion(volume::Volume& outputVolume,
            const volume::Volume& conservedVolume)
        {
            if (!left || left->getNumberOfXCells() != conservedVolume.getNumberOfXCells()) {
                size_t nx = conservedVolume.getNumberOfXCells();
                size_t ny = conservedVolume.getNumberOfYCells();
                size_t nz = conservedVolume.getNumberOfZCells();

                size_t gcx = conservedVolume.getNumberOfXGhostCells();
                size_t gcy = conservedVolume.getNumberOfYGhostCells();
                size_t gcz = conservedVolume.getNumberOfZGhostCells();



                left = volumeFactory.createConservedVolume(nx, ny, nz, gcx);
                right = volumeFactory.createConservedVolume(nx, ny, nz, gcx);
                entropyVariables = volumeFactory.createConservedVolume(nx, ny, nz, gcx);

            }
            typename Equation::ConstViews conservedView(conservedVolume);
            typename Equation::Views entropyVariablesView(*entropyVariables);
            const size_t size = conservedVolume.getTotalNumberOfXCells()
                * conservedVolume.getTotalNumberOfYCells()
                * conservedVolume.getTotalNumberOfZCells();

            const size_t blockSize = 512;
            computeEntropyVariables<Equation> << <(size + blockSize - 1) / blockSize, blockSize >> >(equation, entropyVariablesView, conservedView);


            for (int direction = 0; direction < outputVolume.getDimensions(); ++direction) {
                const ivec3 directionVector(direction == 0, direction == 1, direction == 2);
                reconstruction->performReconstruction(*entropyVariables, direction, 0, *left, *right);

                typename Equation::ConstViews leftView(*left);
                typename Equation::ConstViews rightView(*right);
                typename Equation::ConstViews conservedView(conservedVolume);
                typename Equation::Views outputView(outputVolume);
                const size_t ngc = getNumberOfGhostCells();
                
                const size_t nx = conservedVolume.getTotalNumberOfXCells();
                const size_t ny = conservedVolume.getTotalNumberOfYCells();
                const size_t nz = conservedVolume.getTotalNumberOfZCells();

                const ivec3 start = directionVector;
                const ivec3 end = ivec3(nx, ny, nz) - directionVector;




                const ivec3 numberOfCellsPerDimension = end - start;

                const size_t totalNumberOfCells = size_t(numberOfCellsPerDimension.x) *
                    size_t(numberOfCellsPerDimension.y) *
                    size_t(numberOfCellsPerDimension.z);

               
                multiplyDiffusionMatrix<Equation, DiffusionMatrix>
                    << <(totalNumberOfCells + blockSize - 1) / blockSize, blockSize >> >(
                        equation, outputView,
                        leftView,
                        rightView,
                        conservedView,
                        numberOfCellsPerDimension.x,
                        numberOfCellsPerDimension.y,
                        numberOfCellsPerDimension.z,
                        directionVector
                        );
            }
        }

        template<class Equation, class DiffusionMatrix>
        size_t TecnoDiffusionCUDA<Equation, DiffusionMatrix>::getNumberOfGhostCells() const {
            return reconstruction->getNumberOfGhostCells();
        }

        template class TecnoDiffusionCUDA<::alsfvm::equation::burgers::Burgers, ::alsfvm::diffusion::RoeMatrix<equation::burgers::Burgers> >;
    }
}
